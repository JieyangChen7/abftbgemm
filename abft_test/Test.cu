#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <cstdint>
#include "./abft_checker.h"

void MaxtrixRandom(float *A, int64_t num_batches, int64_t stride, int64_t ld, int64_t row, int64_t col);
void outputChk(float *A, int64_t nb, int64_t ld, int64_t stride, int64_t row, int64_t col);

void abftbgemm(int64_t m, int64_t n, int64_t k, float alpha,
    float *dA, int64_t ldda, int64_t stridea, 
    float *dB, int64_t lddb, int64_t strideb, float beta,
    float *dC, int64_t lddc, int64_t stridec,
    float *dA_colchk, int64_t ldda_colchk, float *dA_rowchk, int64_t ldda_rowchk,
    float *dA_colchk_r, int64_t ldda_colchk_r, float *dA_rowchk_r, int64_t ldda_rowchk_r,
    float *dB_colchk, int64_t lddb_colchk, float *dB_rowchk, int64_t lddb_rowchk,    
    float *dB_colchk_r, int64_t lddb_colchk_r, float *dB_rowchk_r, int64_t lddb_rowchk_r,
    float *dC_colchk, int64_t lddc_colchk, float *dC_rowchk, int64_t lddc_rowchk,
    float *dC_colchk_r, int64_t lddc_colchk_r, float *dC_rowchk_r, int64_t lddc_rowchk_r,
    float *chk_v_a, float *chk_v_b, int64_t ld_chk_v,
    int64_t num_batches,
    bool COL_FT, bool ROW_FT, bool DEBUG, bool CHECK_BEFORE, bool CHECK_AFTER){
    
    std::cout << "Using abftbgemm-at::float function." << std::endl;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipblasSetStream(handle, stream1);

    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;

    float falpha = 1;
    float fbeta = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float t, t1, t_Achk, t_Bchk;
    bool DEBUG_GEMM = true;

    if (DEBUG_GEMM) hipEventRecord(start, stream1);
    if(transA == HIPBLAS_OP_N){
        hipblasSgemmStridedBatched(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 2, k, m,
        &alpha, chk_v_a, ld_chk_v, 0,
        dA, ldda, stridea, &fbeta,
        dA_colchk, ldda_colchk, (2*k),
        num_batches);
        // std::cout << "  Output dA_colchk: " << std::endl;
        // outputChk(dA_colchk, num_batches, ldda_colchk, (2*k), 2, k);
    }
    else{
        hipblasSgemmStridedBatched(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_T, k, 2, m,
        &falpha, dA, ldda, stridea,
        chk_v_a, ld_chk_v, 0, &fbeta,
        dA_rowchk, ldda_rowchk, (2*k),
        num_batches);
        // std::cout << "  Output dA_rowchk: " << std::endl;
        // outputChk(dA_rowchk, num_batches, ldda_rowchk, (2*k), k, 2);
    }
    if (DEBUG_GEMM) {
        hipEventRecord(stop, stream1);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t_Achk, start, stop);
        // printf("dA_chk_gemm: %f (%f)(%f)\n", t, (double)num_batches*m*2*k*2/t/1e6, (double)num_batches*(2*k+2*m+k*m)/t/1e6);
    }

    //std::cout << "  Get dB_chk: " << std::endl;
    if (DEBUG_GEMM) hipEventRecord(start, stream1);
    if (transB == HIPBLAS_OP_N){
        hipblasSgemmStridedBatched(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_T, k, 2, n,
        &alpha, dB, lddb, strideb,
        chk_v_b, ld_chk_v, 0, &fbeta,
        dB_rowchk, lddb_rowchk, (2*k),
        num_batches);
        // std::cout << " Output dB_rowchk: " << std::endl;
        // outputChk(dB_rowchk, num_batches,lddb_rowchk, (2*k), k, 2);
    }
    else{
        hipblasSgemmStridedBatched(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 2, k, n,
        &alpha, chk_v_b, ld_chk_v, 0,
        dB, lddb, strideb, &fbeta,
        dB_colchk, lddb_colchk, (2*k),
        num_batches);
        // std::cout << " Output dB_colchk: " << std::endl;
        // outputMatrixChk(dB_colchk, lddb_colchk, (2*k), num_batches, 2, k);
    }
    if (DEBUG_GEMM) {
        hipEventRecord(stop, stream1);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t_Bchk, start, stop);
        // printf("dB_chk_gemm: %f (%f)(%f)(%f)\n", t1, t1/t, (double)num_batches*2*n*k*2/t1/1e6, (double)num_batches*(2*k+k*n+2*n)/t1/1e6);
    }

    falpha = alpha;
    fbeta = beta;

    // number of row and col of B stored in memory(no trans operation)
    int64_t mem_row = 0;
    int64_t mem_col = 0;

    // --check before beginning-- //
    std::cout << "-----Check Before Beginning------" << std::endl;
    if (COL_FT && CHECK_BEFORE) {
      // number of row and col of A stored in memory(no trans operation)
      if (transA == HIPBLAS_OP_N) {
        mem_row = m;
        mem_col = k;
        if (DEBUG) printf("abftgemm-before-check-A-col\n");
        abft_checker_colchk(handle, transA, transB,
                              dA, ldda, mem_row, mem_col, stridea,
                              dA_colchk,   ldda_colchk,
                              dA_colchk_r, ldda_colchk_r,
                              chk_v_a,       ld_chk_v,
                              DEBUG,
                              stream1,
                              num_batches);
      }
      else if (transA == HIPBLAS_OP_T || transA == HIPBLAS_OP_C) {
        mem_row = k;
        mem_col = m;
        if (DEBUG) printf("dgemm-before-check-A-row\n");
        abft_checker_rowchk(handle, transA, transB,
                              dA, ldda, mem_row, mem_col, stridea,
                              dA_rowchk,   ldda_rowchk,
                              dA_rowchk_r, ldda_rowchk_r,
                              chk_v_a,       ld_chk_v,
                              DEBUG,
                              stream1,
                              num_batches);
      }
      mem_row = m;
      mem_col = n;
      if (DEBUG) printf("abftgemm-before-check-C-col\n");
      abft_checker_colchk(handle, transA, transB,
                              dC, lddc, mem_row, mem_col, stridec,
                              dC_colchk,   lddc_colchk,
                              dC_colchk_r, lddc_colchk_r,
                              chk_v_a,       ld_chk_v,
                              DEBUG,
                              stream1,
                              num_batches);

    }
    if (ROW_FT && CHECK_BEFORE)	{
      //verify B before use
      if (transB == HIPBLAS_OP_N) {
        mem_row = k;
        mem_col = n;
        if (DEBUG) printf("dgemm-before-check-B-row\n");
        abft_checker_rowchk(handle, transA, transB,
                                dB, lddb, mem_row, mem_col, strideb,
                                dB_rowchk,   lddb_rowchk,
                                dB_rowchk_r, lddb_rowchk_r,
                                chk_v_b,       ld_chk_v,
                                DEBUG,
                                stream1,
                                num_batches);

      }
      else if (transB == HIPBLAS_OP_T || transB == HIPBLAS_OP_C) {
        mem_row = n;
        mem_col = k;
        if (DEBUG) printf("dgemm-before-check-B-col\n");
        abft_checker_colchk(handle, transA, transB,
                                dB, lddb, mem_row, mem_col, strideb,
                                dB_colchk,   lddb_colchk,
                                dB_colchk_r, lddb_colchk_r,
                                chk_v_b,       ld_chk_v,
                                DEBUG,
                                stream1,
                                num_batches);
      }
      mem_row = m;
      mem_col = n;
      if (DEBUG) printf("dgemm-before-check-C-row\n");
      abft_checker_rowchk(handle, transA, transB,
                              dC, lddc, mem_row, mem_col, stridec,
                              dC_rowchk,   lddc_rowchk,
                              dC_rowchk_r, lddc_rowchk_r,
                              chk_v_b,       ld_chk_v,
                              DEBUG,
                              stream1,
                              num_batches);
    }

    std::cout << "-----Begin.------" << std::endl;

    if (DEBUG_GEMM) hipEventRecord(start, stream1);
    std::cout<<"A*B=C." << std::endl;
    hipblasSgemmStridedBatched(
        handle, transA, transB, m, n, k,
        &falpha, dA, ldda, stridea,
        dB, lddb, strideb, &fbeta,
        dC, lddc, stridec,
        num_batches);
    // std::cout << "Output dC: " << std::endl;
    // outputMatrix(dC, lddc, stridec, num_batches, m, n);
    
    if (DEBUG_GEMM) {
      hipEventRecord(stop, stream1);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t, start, stop);
      printf("  gemm: %f (%f)(%f)\n", t, (double)num_batches*m*n*k*2/t/1e6, (double)num_batches*(m*k+k*n+m*n)/t/1e6);
      printf("dA_chk_gemm: %f (%f)(%f)(%f)\n", t_Achk, t_Achk/t, (double)num_batches*m*2*k*2/t_Achk/1e6, (double)num_batches*(2*k+2*m+k*m)/t_Achk/1e6);
      printf("dB_chk_gemm: %f (%f)(%f)(%f)\n", t_Bchk, t_Bchk/t, (double)num_batches*2*n*k*2/t_Bchk/1e6, (double)num_batches*(2*k+k*n+2*n)/t_Bchk/1e6);
    }

    if (DEBUG_GEMM) hipEventRecord(start, stream1);
    if(COL_FT){
      //std::cout << "  COL_FT" << std::endl;
      if (transA == HIPBLAS_OP_N) {
        std::cout << "dA_colchk * dB = dC_colchk" << std::endl;
        hipblasSgemmStridedBatched(
            handle, transA, transB, 2, n, k,
            &falpha, dA_colchk, ldda_colchk, k*2,
            dB, lddb, strideb, &fbeta,
            dC_colchk, lddc_colchk, n*2,
            num_batches);
      }
      else{
        std::cout << "dB * dA_rowchk = dC_colchk" << std::endl;
        hipblasSgemmStridedBatched(
            handle, transA, transB, 2, n, k,
            &falpha, dA_rowchk, ldda_rowchk, k*2,
            dB, lddb, strideb, &fbeta,
            dC_colchk, lddc_colchk, n*2,
            num_batches);
      }
      // std::cout << "Output dC_colchk: " << std::endl;
      // outputMatrixChk(dC_colchk, ldda_colchk, n*2, num_batches, 2, n);
    }
    if (DEBUG_GEMM) {
        hipEventRecord(stop, stream1);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t1, start, stop);
        printf("  gemm-col-ft: %f (%f)(%f)(%f)\n", t1, t1/t, (double)num_batches*2*n*k*2/t1/1e6, (double)num_batches*(2*k+k*n+2*n)/t1/1e6);
    }

    if (DEBUG_GEMM) hipEventRecord(start, stream1);
    if (ROW_FT) {
        //std::cout << "  ROW_FT" << std::endl;
        if (transB == HIPBLAS_OP_N) {
          std::cout << "dA * dB_rowchk = dC_rowlchk" << std::endl;
          //we can further work on this to support trans A.
          hipblasSgemmStridedBatched(
            handle, transA, transB, m, 2, k,
            &falpha, dA, ldda, stridea,
            dB_rowchk, lddb_rowchk, k*2, &fbeta,
            dC_rowchk, lddc_rowchk, m*2,
            num_batches);
        }
        else{
          std::cout << "dB_colchk * dA = dC_rowlchk" << std::endl;
          hipblasSgemmStridedBatched(
            handle, transA, transB, m, 2, k,
            &falpha, dA, ldda, stridea,
            dB_colchk, lddb_colchk, k*2, &fbeta,
            dC_rowchk, lddc_rowchk, m*2,
            num_batches);
        }
        // std::cout << "Output dC_rowchk: " << std::endl;
        // outputMatrixChk(dC_rowchk,lddc_rowchk, m*2, num_batches, m, 2);
    }
    if (DEBUG_GEMM) {
      hipEventRecord(stop, stream1);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t1, start, stop);
      printf("  gemm-row-ft: %f (%f)(%f)(%f)\n", t1, t1/t, (double)num_batches*m*2*k*2/t1/1e6, (double)num_batches*(m*k+k*2+m*2)/t1/1e6);
    }

    // --- check check-sum of C---//
    std::cout << "------Check check-sum-------" << std::endl;
    if (DEBUG_GEMM) hipEventRecord(start, stream1);
    if (COL_FT && CHECK_AFTER) {
      mem_row = m;
      mem_col = n;
      if (DEBUG) printf("dgemm-after-check-C-col\n");
      abft_checker_colchk(handle, transA, transB,
                              dC, lddc, mem_row, mem_col, stridec,
                              dC_colchk,   lddc_colchk,
                              dC_colchk_r, lddc_colchk_r,
                              chk_v_a,       ld_chk_v,
                              DEBUG,
                              stream1,
                              num_batches);
    }

    if (DEBUG_GEMM) {
      hipEventRecord(stop, stream1);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t1, start, stop);
      printf("gemm-col-chk: %f (%f)(%f)(%f)\n", t1, t1/t, (double)(num_batches)*2*n*m*2/t1/1e6, (double)num_batches*(m*n+2*m+2*n)/t1/1e6);
    }

    if (DEBUG_GEMM) hipEventRecord(start, stream1);
    if (ROW_FT && CHECK_AFTER) {
      mem_row = m;
      mem_col = n;
      if (DEBUG) printf("dgemm-after-check-C-row\n");
      abft_checker_rowchk(handle, transA, transB,
                              dC, lddc, mem_row, mem_col, stridec,
                              dC_rowchk,   lddc_rowchk,
                              dC_rowchk_r, lddc_rowchk_r,
                              chk_v_b,       ld_chk_v,
                              DEBUG,
                              stream1,
                              num_batches);

    }

    if (DEBUG_GEMM) {
        hipEventRecord(stop, stream1);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t1, start, stop);
        printf("gemm-row-chk: %f (%f)(%f)(%f)\n", t1, t1/t, (double)(num_batches)*m*2*n*2/t1/1e6, (double)num_batches*(m*n+2*n+2*m)/t1/1e6);
    }

    return 0;

}


int main(){
    float *A, *B;
    float *dA, *dB, *dC;

    int64_t m = 72;
    int64_t n = 72;
    int64_t k = 64;
    int64_t num_batches = 96;

    size_t size = num_batches * m * k * sizeof(float);
    hipMalloc((void **)&dA, size);
    A = (float *)malloc(size);
    MaxtrixRandom(A, num_batches, m*k, m, m, k);
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    // printf("dA: \n");
    // outputChk(dA, num_batches, m, m*k, m, k); 

    size = num_batches * k * n * sizeof(float);
    hipMalloc((void **)&dB, size);
    B = (float *)malloc(size);
    MaxtrixRandom(B, num_batches, k*n, k, k, n);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);
    // printf("dB: \n");
    // outputChk(dB, num_batches, k, n*k, k, n);
    
    size = num_batches * m * n * sizeof(float);
    hipMalloc((void **)&dC, size);
    hipMemset(dC, 0, (num_batches * m * n * sizeof(float)));

    int64_t ldda_colchk = 2;
    int64_t ldda_colchk_r = 2;
    int64_t ldda_rowchk = k;
    int64_t ldda_rowchk_r = k;

    int64_t lddb_rowchk = k;
    int64_t lddb_rowchk_r = k;
    int64_t lddb_colchk = 2;
    int64_t lddb_colchk_r = 2;

    int64_t lddc_colchk = 2;
    int64_t lddc_colchk_r = 2;
    int64_t lddc_rowchk = m;
    int64_t lddc_rowchk_r = m;
    int64_t ld_chk_v = 2;

    float *dA_colchk, *dA_rowchk, *dA_colchk_r, *dA_rowchk_r;
    float *dB_colchk, *dB_rowchk, *dB_colchk_r, *dB_rowchk_r;
    float *dC_colchk, *dC_rowchk, *dC_colchk_r, *dC_rowchk_r;
    float *chk_v_a;
    float *chk_v_b;

    size = (2*num_batches) * k * sizeof(float);
    hipMalloc((void**)&dA_colchk, size);
    hipMemset(dA_colchk, 0, size);
    hipMalloc((void**)&dA_colchk_r, size);
    hipMemset(dA_colchk_r, 0, size);

    hipMalloc((void**)&dA_rowchk, size);
    hipMemset(dA_rowchk, 0, size);
    hipMalloc((void**)&dA_rowchk_r, size);
    hipMemset(dA_rowchk_r, 0, size);
    //std::cout << "  finish dA." << std::endl;
    
    hipMalloc((void**)&dB_colchk, size);
    hipMemset(dB_colchk, 0, size);
    hipMalloc((void**)&dB_colchk_r, size);
    hipMemset(dB_colchk_r, 0, size);
    
    hipMalloc((void**)&dB_rowchk, size);
    hipMemset(dB_rowchk, 0, size);
    hipMalloc((void**)&dB_rowchk_r, size);
    hipMemset(dB_rowchk_r, 0, size);
    //std::cout << "  finish dB." << std::endl;

    size = (2*num_batches) * n * sizeof(float);
    hipMalloc((void**)&dC_colchk, size);
    hipMemset(dC_colchk, 0, size);
    hipMalloc((void**)&dC_colchk_r, size);
    hipMemset(dC_colchk_r, 0, size);
    
    size = (2*num_batches) * m * sizeof(float);
    hipMalloc((void**)&dC_rowchk, size);
    hipMemset(dC_rowchk, 0, size);
    hipMalloc((void**)&dC_rowchk_r, size);
    hipMemset(dC_rowchk_r, 0, size);

    int64_t len = m;
    size = 2 * len * sizeof(float);
    hipMalloc((void**)&chk_v_a, size);
    // std::cout << "  assign values to chk_v_a." << std::endl;
    float *h_matrix;
    h_matrix = (float *)malloc(size);
    int idx = 0;
    for(int i = 0; i < len; i++){
        idx = i*ld_chk_v;
        h_matrix[idx] = float(1);
        h_matrix[idx+1] = float(i+1);
    }
    hipMemcpy(chk_v_a, h_matrix, size, hipMemcpyHostToDevice);
    // std::cout << "chk_v_a: " << std::endl;
    // outputChk(chk_v_a, 1, ld_chk_v, 0, 2, m);
    free(h_matrix);

    len = n;
    size = 2 * len * sizeof(float);
    hipMalloc((void**)&chk_v_b, size);
    // std::cout << "  assign values to chk_v_b." << std::endl;
    h_matrix = (float *)malloc(size);
    idx = 0;
    for(int i = 0; i < len; i++){
        idx = i*ld_chk_v;
        h_matrix[idx] = float(1);
        h_matrix[idx+1] = float(i+1);
    }
    hipMemcpy(chk_v_b, h_matrix, size, hipMemcpyHostToDevice);
    // std::cout << "chk_v_b: " << std::endl;
    // outputChk(chk_v_a, 1, ld_chk_v, 0, 2, len);
    free(h_matrix);
    //std::cout << "  finish chk_v." << std::endl;

    bool COL_FT = true;
    bool ROW_FT = true;
    bool DEBUG = true;
    bool CHECK_BEFORE = true;
    bool CHECK_AFTER = true;

    float alpha = 1;
    float beta = 0;
    int64_t stridea = m*k;
    int64_t strideb = n*k;
    int64_t stridec = m*n;
    int64_t ldda = m;
    int64_t lddb = k;
    int64_t lddc = m;


    abftbgemm(m, n, k,
        alpha, dA, ldda, stridea,
        dB, lddb, strideb, beta,
        dC, lddc, stridec,
        dA_colchk, ldda_colchk,
        dA_rowchk, ldda_rowchk,
        dA_colchk_r, ldda_colchk_r,
        dA_rowchk_r, ldda_rowchk_r,
        dB_colchk, lddb_colchk,
        dB_rowchk, lddb_rowchk,
        dB_colchk_r, lddb_colchk_r,
        dB_rowchk_r, lddb_rowchk_r,
        dC_colchk, lddc_colchk,
        dC_rowchk, lddc_rowchk,
        dC_colchk_r, lddc_colchk_r,
        dC_rowchk_r, lddc_rowchk_r,
        chk_v_a, chk_v_b, ld_chk_v,
        num_batches,
        COL_FT,ROW_FT,DEBUG,CHECK_BEFORE,CHECK_AFTER);

    hipFree(dA_colchk);
    hipFree(dA_rowchk);
    hipFree(dA_colchk_r);
    hipFree(dA_rowchk_r);
    hipFree(dB_colchk);
    hipFree(dB_rowchk);
    hipFree(dB_colchk_r);
    hipFree(dB_rowchk_r);
    hipFree(dC_colchk);
    hipFree(dC_rowchk);
    hipFree(dC_colchk_r);
    hipFree(dC_rowchk_r);
    hipFree(chk_v_a);
    hipFree(chk_v_b);

    return 0;
}


void MaxtrixRandom(float *A, int64_t num_batches, int64_t stride, int64_t ld, int64_t row, int64_t col){
  for(int num = 0; num < num_batches; num++){
    for (int r = 0; r < row; r++){
      for (int c = 0; c < col; c++){
        // A[num*stride + c*ld + r] = ((float)rand() / RAND_MAX);
        // (half)((float)(rand()) / (float)(rand()));
        A[num*stride + c*ld + r] = 1;
      }
    }
  }
}

void outputChk(float *A, int64_t nb, int64_t ld, int64_t stride, int64_t row, int64_t col){
  size_t size = nb * (row * col) * sizeof(float);
  float *tensor;
  tensor = (float *)malloc(size);
  hipMemcpy(tensor, A, size, hipMemcpyDeviceToHost);
  for(int i = 0; i < nb; i++){
    printf("[ \n");
    for(int r = 0; r < row; r++){
      for(int c = 0; c < col; c++){
        printf("%.6f", float(tensor[i*stride + c*ld + r]));
        printf(", ");
      }
      printf("\n");
    }
    printf("]\n");
  }
  free(tensor);
}